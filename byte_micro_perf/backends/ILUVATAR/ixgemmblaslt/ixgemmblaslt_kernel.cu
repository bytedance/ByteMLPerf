#include <iostream>
#include "ixgemmblaslt.hpp"

//#define CAL_TFLOPS_TEST

gemm_kernel_param gemm_kernel_init()
{
  hipblasLtHandle_t lt_handle = nullptr;
  checkBlasStatus(hipblasLtCreate(&(lt_handle)));

  hipblasLtMatmulDesc_t op_desc = nullptr;
#ifdef __ILUVATAR__
    hipDataType compute_type = HIP_R_32I;
#else
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32I;
#endif
  //hipDataType scale_type = HIP_R_32I;
  hipDataType scale_type = HIP_R_32F;
  hipblasOperation_t op_trans_a = HIPBLAS_OP_N;
  hipblasOperation_t op_trans_b = HIPBLAS_OP_N;
#ifdef __ILUVATAR__
    checkBlasStatus(hipblasLtMatmulDescCreate(&op_desc, compute_type));
#else
    checkBlasStatus(hipblasLtMatmulDescCreate(&op_desc, compute_type, scale_type));
#endif
  checkBlasStatus(hipblasLtMatmulDescSetAttribute(op_desc, CUBLASLT_MATMUL_DESC_SCALE_TYPE, &scale_type, sizeof(scale_type)));
  checkBlasStatus(hipblasLtMatmulDescSetAttribute(op_desc, HIPBLASLT_MATMUL_DESC_TRANSA, &op_trans_a, sizeof(op_trans_a)));
  checkBlasStatus(hipblasLtMatmulDescSetAttribute(op_desc, HIPBLASLT_MATMUL_DESC_TRANSB, &op_trans_b, sizeof(op_trans_b)));

  gemm_kernel_param ins;
  ins.lt_handle = reinterpret_cast<uintptr_t>(lt_handle);
  ins.op_desc = reinterpret_cast<uintptr_t>(op_desc);

  return ins;
}

void gemm_kernel_run(gemm_kernel_param ins, char *d_A, char *d_B, char *d_C, const int M, const int N, const int K)
{
  float alpha_int8 = 1.0;
  float beta_int8 = 0.0;
  hipDataType ab_type = HIP_R_8I;
  //hipDataType c_type = HIP_R_32I;
  hipDataType c_type = HIP_R_8I;
  hipblasLtMatrixLayout_t a_desc = nullptr, b_desc = nullptr, c_desc = nullptr;

  hipblasLtHandle_t lt_handle = reinterpret_cast<hipblasLtHandle_t>(reinterpret_cast<uintptr_t *>(ins.lt_handle));
  hipblasLtMatmulDesc_t op_desc = reinterpret_cast<hipblasLtMatmulDesc_t>(reinterpret_cast<uintptr_t *>(ins.op_desc));

  checkBlasStatus(hipblasLtMatrixLayoutCreate(&a_desc, ab_type, K, M, K));
  checkBlasStatus(hipblasLtMatrixLayoutCreate(&b_desc, ab_type, N, K, N));
  checkBlasStatus(hipblasLtMatrixLayoutCreate(&c_desc, c_type, N, M, N));

#ifdef CAL_TFLOPS_TEST
  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();
#endif

  checkBlasStatus(hipblasLtMatmul(lt_handle, op_desc, &alpha_int8, d_B, b_desc, d_A, a_desc, &beta_int8, d_C, c_desc, d_C, c_desc, nullptr, nullptr, 0, nullptr));

#ifdef CAL_TFLOPS_TEST  
  hipDeviceSynchronize();
  auto stop = std::chrono::steady_clock::now();

  std::chrono::duration<double, std::milli> dur_ms = stop - start;
  double elapse = dur_ms.count();
  double tflops = 1e-9 * 2.0f * M * N * K;
  printf("\n---------------elapse: %lf ms, TOPs: %lf\n\n", elapse, tflops / elapse);
#endif
}

void gemm_kernel_release(gemm_kernel_param ins)
{
  hipblasLtHandle_t lt_handle = reinterpret_cast<hipblasLtHandle_t>(reinterpret_cast<uintptr_t *>(ins.lt_handle));
  hipblasLtDestroy(lt_handle);
}

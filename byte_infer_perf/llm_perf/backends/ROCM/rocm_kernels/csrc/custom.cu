#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>

// declare templates for front (cpp) and back (cuda) sides of function:
// template <typename T>

// void LLGemm_Silu(void* in_a, void* in_b, void* out_c, const int M, const int K,
//                  hipStream_t stream, const int rows_per_block);
// void LLMM_Silu(at::Tensor& in_a, at::Tensor& in_b, at::Tensor& out_c,
//                const int64_t rows_per_block) {
//   auto M = in_a.size(0);
//   auto K = in_a.size(1);
//   LLGemm_Silu(in_a.data_ptr(), in_b.data_ptr(), out_c.data_ptr(), M, K,
//               at::cuda::getCurrentCUDAStream(), rows_per_block);
// }

void LLGemm1(void* in_a, void* in_b, void* out_c, const int M, const int K,
             hipStream_t stream, const int rows_per_block);

// template <typename T>
void LLMM1(at::Tensor& in_a, at::Tensor& in_b, at::Tensor& out_c,
           const int64_t rows_per_block) {
  auto M = in_a.size(0);
  auto K = in_a.size(1);
  // if (N != in_b.numel())
  //         throw std::invalid_argument("Size mismatch A.numel(): " +
  //         std::to_string(in_a.numel())
  //                           + ", B.numel(): " +
  //                           std::to_string(in_b.numel()));

  // out_c.resize_({N});

  // call the kernel function...
  LLGemm1(in_a.data_ptr(), in_b.data_ptr(), out_c.data_ptr(), M, K,
          at::cuda::getCurrentCUDAStream(), rows_per_block);
}

void wvSpltK_(void* in_a, void* in_b, void* out_c, const int M, const int K,
              const int N, hipStream_t stream, const int CuCount);

void wvSpltK(at::Tensor& in_a, at::Tensor& in_b, at::Tensor& out_c,
             const int64_t N_in, const int64_t CuCount) {
  auto M = in_a.size(0);
  auto K = in_a.size(1);
  int N = N_in;
  wvSpltK_(in_a.data_ptr(), in_b.data_ptr(), out_c.data_ptr(), M, K, N,
           at::cuda::getCurrentCUDAStream(), CuCount);
}

void LLGemmZZ(void* in_a, void* in_b, void* out_c, const int M, const int K,
              hipStream_t stream, const int solidx);

void LLZZ(at::Tensor in_a, at::Tensor in_b, at::Tensor out_c,
          const int64_t solidx = 0) {
  auto M = in_a.size(0);
  auto K = in_a.size(1);

  LLGemmZZ(in_a.data_ptr(), in_b.data_ptr(), out_c.data_ptr(), M, K,
           at::cuda::getCurrentCUDAStream(), solidx);
}
// instantiate the CPP template for T=float:
// template void AddGPU<float>(at::Tensor in_a, at::Tensor in_b, at::Tensor
// out_c);

void MMGPUKernel(float* in_a, float* in_b, float* out_c, int numARows,
                 int numAColumns, int numBRows, int numBColumns, int numCRows,
                 int numCColumns, hipStream_t stream);

void MMCustomGPU(at::Tensor& in_a, at::Tensor& in_b, at::Tensor& out_c) {
  auto matA_sizes{in_a.sizes()};
  auto matB_sizes{in_b.sizes()};
  auto matO_sizes{out_c.sizes()};
  MMGPUKernel(in_a.data_ptr<float>(), in_b.data_ptr<float>(),
              out_c.data_ptr<float>(), matA_sizes[0], matA_sizes[1],
              matB_sizes[0], matB_sizes[1], matO_sizes[0], matO_sizes[1],
              at::cuda::getCurrentCUDAStream());
}

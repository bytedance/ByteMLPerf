#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "dispatch_utils.h"
// #include "attention/attention_dtypes.h"
#ifndef USE_ROCM
  #include <hip/hip_bf16.h>
  #include <hip/hip_fp16.h>
  #include <cub/util_type.cuh>
  #include <hipcub/hipcub.hpp>
#else
  #include <hip/hip_bf16.h>
  #include <hip/hip_fp16.h>
  #include <hipcub/util_type.hpp>
  #include <hipcub/hipcub.hpp>
  // #include "quantization/fp8/amd/hip_float8.h"
  // #include "quantization/fp8/amd/quant_utils.cuh"

using __hip_bfloat16 = __hip_bfloat16;
using __hip_bfloat162 = __hip_bfloat162;
#endif


namespace vllm {

template <typename scalar_t>
struct __align__(16) vec8_t {
  scalar_t x, y, z, w, u, v, s, t;

  __device__ vec8_t() : x(0), y(0), z(0), w(0), u(0), v(0), s(0), t(0) {}
  __device__ vec8_t(scalar_t x, scalar_t y, scalar_t z, scalar_t w, scalar_t u,
                    scalar_t v, scalar_t s, scalar_t t)
      : x(x), y(y), z(z), w(w), u(u), v(v), s(s), t(t) {}

  __device__ vec8_t operator*(const vec8_t& other) const {
    return vec8_t(x * other.x, y * other.y, z * other.z, w * other.w,
                  u * other.u, v * other.v, s * other.s, t * other.t);
  }

  __device__ vec8_t operator*(const float& scale) const {
    return vec8_t(x * scale, y * scale, z * scale, w * scale, u * scale,
                  v * scale, s * scale, t * scale);
  }

  __device__ vec8_t operator+(const vec8_t& other) const {
    return vec8_t(x + other.x, y + other.y, z + other.z, w + other.w,
                  u + other.u, v + other.v, s + other.s, t + other.t);
  }

  __device__ void operator+=(const vec8_t& other) {
    x += other.x;
    y += other.y;
    z += other.z;
    w += other.w;
    u += other.u;
    v += other.v;
    s += other.s;
    t += other.t;
  }

  __device__ scalar_t sum() const { return x + y + z + w + u + v + s + t; }
};

// TODO(woosuk): Further optimize this kernel.
template <typename scalar_t>
__global__ void rms_norm_kernel(
    scalar_t* __restrict__ out,           // [..., hidden_size]
    const scalar_t* __restrict__ input,   // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;

  vec8_t<scalar_t> v8_variance = {0, 0, 0, 0, 0, 0, 0, 0};

  vec8_t<scalar_t>* vectorized_out = reinterpret_cast<vec8_t<scalar_t>*>(out);
  vec8_t<scalar_t> const* vectorized_in =
      reinterpret_cast<vec8_t<scalar_t> const*>(input);
  vec8_t<scalar_t> const* vectorized_weight =
      reinterpret_cast<vec8_t<scalar_t> const*>(weight);
  const int vec_hidden_size = hidden_size >> 3;

  // Compute variance. Be careful, hidden_size should multiple of 4.
  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    vec8_t<scalar_t> x = vectorized_in[blockIdx.x * vec_hidden_size + idx];
    v8_variance += x * x;
  }
  float v8_variance_sum = v8_variance.sum();

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  float variance =
      BlockReduce(reduceStore).Reduce(v8_variance_sum, hipcub::Sum{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    vec8_t<scalar_t> v8_in = vectorized_in[blockIdx.x * vec_hidden_size + idx];
    vec8_t<scalar_t> v8_w = vectorized_weight[idx];
    vectorized_out[blockIdx.x * vec_hidden_size + idx] =
        v8_in * s_variance * v8_w;
  }
}

// template <typename scalar_t>
// __global__ void scaled_rms_norm_kernel(
//     c10::Float8_e4m3fnuz* __restrict__ out,  // [..., hidden_size]
//     const scalar_t* __restrict__ input,      // [..., hidden_size]
//     const scalar_t* __restrict__ weight,     // [hidden_size]
//     const float scale, const float epsilon, const int num_tokens,
//     const int hidden_size) {
//   __shared__ float s_variance;
//   float variance = 0.0f;

//   for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
//     const float x = (float)input[blockIdx.x * hidden_size + idx];
//     variance += x * x;
//   }

//   using BlockReduce = hipcub::BlockReduce<float, 1024>;
//   __shared__ typename BlockReduce::TempStorage reduceStore;
//   variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

//   if (threadIdx.x == 0) {
//     s_variance = rsqrtf(variance / hidden_size + epsilon);
//   }
//   __syncthreads();

//   for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
//     float x = (float)input[blockIdx.x * hidden_size + idx];
//     float r = (x * s_variance) * weight[idx] * scale;
//     out[blockIdx.x * hidden_size + idx] = c10::Float8_e4m3fnuz(
//         hip_fp8(r).data, c10::Float8_e4m3fnuz::from_bits());
//   }
// }

/* Converter structs for the conversion from torch types to HIP/CUDA types,
   and the associated type conversions within HIP/CUDA. These helpers need
   to be implemented for now because the relevant type conversion
   operators/constructors are not consistently implemented by HIP/CUDA, so
   a generic conversion via type casts cannot be implemented.

   Each struct should have the member static constexpr bool `exists`:
   If false, the optimized kernel is not used for the corresponding torch type.
   If true, the struct should be fully defined as shown in the examples below.
 */
template <typename torch_type>
struct _typeConvert {
  static constexpr bool exists = false;
};

#if defined(USE_ROCM) || (defined(CUDA_VERSION) && (CUDA_VERSION >= 12000))
// CUDA < 12.0 runs into issues with packed type conversion
template <>
struct _typeConvert<c10::Half> {
  static constexpr bool exists = true;
  using hip_type = __half;
  using packed_hip_type = __half2;

  __device__ static inline float convert(hip_type x) { return __half2float(x); }
  __device__ static inline float2 convert(packed_hip_type x) {
    return __half22float2(x);
  }
  __device__ static inline hip_type convert(float x) {
    return __float2half_rn(x);
  }
  __device__ static inline packed_hip_type convert(float2 x) {
    return __float22half2_rn(x);
  }
};

  #if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
// CUDA_ARCH < 800 does not have BF16 support
// TODO: Add in ROCm support once public headers handle bf16 maturely
template <>
struct _typeConvert<c10::BFloat16> {
  static constexpr bool exists = true;
  using hip_type = __hip_bfloat16;
  using packed_hip_type = __hip_bfloat162;

  __device__ static inline float convert(hip_type x) {
    return __bfloat162float(x);
  }
  __device__ static inline float2 convert(packed_hip_type x) {
    return __bfloat1622float2(x);
  }
  __device__ static inline hip_type convert(float x) {
    return __float2bfloat16(x);
  }
  __device__ static inline packed_hip_type convert(float2 x) {
    return __float22bfloat162_rn(x);
  }
};
  #endif  // defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
#endif    // defined(USE_ROCM) || (defined(CUDA_VERSION) && (CUDA_VERSION >=
          // 12000))

/* Vector POD struct to generate vectorized and packed FP16/BF16 ops
   for appropriate specializations of fused_add_rms_norm_kernel.
   Only functions that are necessary in that kernel are implemented.
   Alignment to 16 bytes is required to use 128-bit global memory ops.
 */
template <typename scalar_t, int width>
struct alignas(16) _f16Vec {
  /* Not theoretically necessary that width is a power of 2 but should
     almost always be the case for optimization purposes */
  static_assert(width > 0 && (width & (width - 1)) == 0,
                "Width is not a positive power of 2!");
  using Converter = _typeConvert<scalar_t>;
  using T1 = typename Converter::hip_type;
  using T2 = typename Converter::packed_hip_type;
  T1 data[width];

  __device__ _f16Vec& operator+=(const _f16Vec<scalar_t, width>& other) {
    if constexpr (width % 2 == 0) {
#pragma unroll
      for (int i = 0; i < width; i += 2) {
        T2 temp{data[i], data[i + 1]};
        temp += T2{other.data[i], other.data[i + 1]};
        data[i] = temp.x;
        data[i + 1] = temp.y;
      }
    } else {
#pragma unroll
      for (int i = 0; i < width; ++i) data[i] += other.data[i];
    }
    return *this;
  }

  __device__ _f16Vec& operator*=(const _f16Vec<scalar_t, width>& other) {
    if constexpr (width % 2 == 0) {
#pragma unroll
      for (int i = 0; i < width; i += 2) {
        T2 temp{data[i], data[i + 1]};
        temp *= T2{other.data[i], other.data[i + 1]};
        data[i] = temp.x;
        data[i + 1] = temp.y;
      }
    } else {
#pragma unroll
      for (int i = 0; i < width; ++i) data[i] *= other.data[i];
    }
    return *this;
  }

  __device__ _f16Vec& operator*=(const float scale) {
    if constexpr (width % 2 == 0) {
#pragma unroll
      for (int i = 0; i < width; i += 2) {
        float2 temp_f = Converter::convert(T2{data[i], data[i + 1]});
        temp_f.x *= scale;
        temp_f.y *= scale;
        T2 temp = Converter::convert(temp_f);
        data[i] = temp.x;
        data[i + 1] = temp.y;
      }
    } else {
#pragma unroll
      for (int i = 0; i < width; ++i) {
        float temp = Converter::convert(data[i]) * scale;
        data[i] = Converter::convert(temp);
      }
    }
    return *this;
  }

  __device__ float sum_squares() const {
    float result = 0.0f;
    if constexpr (width % 2 == 0) {
#pragma unroll
      for (int i = 0; i < width; i += 2) {
        float2 z = Converter::convert(T2{data[i], data[i + 1]});
        result += z.x * z.x + z.y * z.y;
      }
    } else {
#pragma unroll
      for (int i = 0; i < width; ++i) {
        float x = Converter::convert(data[i]);
        result += x * x;
      }
    }
    return result;
  }
};

/* Function specialization in the case of FP16/BF16 tensors.
   Additional optimizations we can make in this case are
   packed and vectorized operations, which help with the
   memory latency bottleneck. */
template <typename scalar_t, int width>
__global__ std::enable_if_t<(width > 0) && _typeConvert<scalar_t>::exists>
fused_add_rms_norm_kernel(
    scalar_t* __restrict__ input,         // [..., hidden_size]
    scalar_t* __restrict__ residual,      // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float epsilon, const int num_tokens, const int hidden_size) {
  // Sanity checks on our vector struct and type-punned pointer arithmetic
  static_assert(std::is_pod_v<_f16Vec<scalar_t, width>>);
  static_assert(sizeof(_f16Vec<scalar_t, width>) == sizeof(scalar_t) * width);

  const int vec_hidden_size = hidden_size / width;
  __shared__ float s_variance;
  float variance = 0.0f;
  /* These and the argument pointers are all declared `restrict` as they are
     not aliased in practice. Argument pointers should not be dereferenced
     in this kernel as that would be undefined behavior */
  auto* __restrict__ input_v =
      reinterpret_cast<_f16Vec<scalar_t, width>*>(input);
  auto* __restrict__ residual_v =
      reinterpret_cast<_f16Vec<scalar_t, width>*>(residual);
  auto* __restrict__ weight_v =
      reinterpret_cast<const _f16Vec<scalar_t, width>*>(weight);

  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    int id = blockIdx.x * vec_hidden_size + idx;
    _f16Vec<scalar_t, width> temp = input_v[id];
    temp += residual_v[id];
    variance += temp.sum_squares();
    residual_v[id] = temp;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
    int id = blockIdx.x * vec_hidden_size + idx;
    _f16Vec<scalar_t, width> temp = residual_v[id];
    temp *= s_variance;
    temp *= weight_v[idx];
    input_v[id] = temp;
  }
}

/* Generic fused_add_rms_norm_kernel
   The width field is not used here but necessary for other specializations.
 */
template <typename scalar_t, int width>
__global__ std::enable_if_t<(width == 0) || !_typeConvert<scalar_t>::exists>
fused_add_rms_norm_kernel(
    scalar_t* __restrict__ input,         // [..., hidden_size]
    scalar_t* __restrict__ residual,      // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    scalar_t z = input[blockIdx.x * hidden_size + idx];
    z += residual[blockIdx.x * hidden_size + idx];
    float x = (float)z;
    variance += x * x;
    residual[blockIdx.x * hidden_size + idx] = z;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)residual[blockIdx.x * hidden_size + idx];
    input[blockIdx.x * hidden_size + idx] =
        ((scalar_t)(x * s_variance)) * weight[idx];
  }
}

/* Function specialization in the case of FP16/BF16 tensors.
   Additional optimizations we can make in this case are
   packed and vectorized operations, which help with the
   memory latency bottleneck. */

// template <>
// struct Vec<c10::Float8_e4m3fnuz, 8> {
//   using Type = uint2;
// };

// template <>
// struct Vec<c10::Half, 8> {
//   using Type = uint4;
// };

// template <>
// struct Vec<c10::BFloat16, 8> {
//   using Type = bf16_8_t;
// };

// template <typename scalar_t, int width>
// __global__ std::enable_if_t<(width > 0) && _typeConvert<scalar_t>::exists>
// scaled_fused_add_rms_norm_kernel(
//     c10::Float8_e4m3fnuz* __restrict__ out,  // [..., hidden_size]
//     scalar_t* __restrict__ input,            // [..., hidden_size]
//     scalar_t* __restrict__ residual,         // [..., hidden_size]
//     const scalar_t* __restrict__ weight,     // [hidden_size]
//     const float epsilon, const float scale, const int num_tokens,
//     const int hidden_size) {
//   using in_v_t = typename Vec<scalar_t, width>::Type;
//   using out_v_t = typename Vec<c10::Float8_e4m3fnuz, width>::Type;
//   // Sanity checks on our vector struct and type-punned pointer arithmetic
//   static_assert(std::is_pod_v<_f16Vec<scalar_t, width>>);
//   static_assert(sizeof(_f16Vec<scalar_t, width>) == sizeof(scalar_t) * width);

//   const int vec_hidden_size = hidden_size / width;
//   __shared__ float s_variance;
//   float variance = 0.0f;
//   /* These and the argument pointers are all declared `restrict` as they are
//      not aliased in practice. Argument pointers should not be dereferenced
//      in this kernel as that would be undefined behavior */
//   auto* __restrict__ out_v = reinterpret_cast<out_v_t*>(out);
//   auto* __restrict__ input_v =
//       reinterpret_cast<_f16Vec<scalar_t, width>*>(input);
//   auto* __restrict__ residual_v =
//       reinterpret_cast<_f16Vec<scalar_t, width>*>(residual);
//   auto* __restrict__ weight_v =
//       reinterpret_cast<const _f16Vec<scalar_t, width>*>(weight);

//   for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
//     int id = blockIdx.x * vec_hidden_size + idx;
//     _f16Vec<scalar_t, width> temp = input_v[id];
//     temp += residual_v[id];
//     variance += temp.sum_squares();
//     residual_v[id] = temp;
//   }

//   using BlockReduce = hipcub::BlockReduce<float, 1024>;
//   __shared__ typename BlockReduce::TempStorage reduceStore;
//   variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

//   if (threadIdx.x == 0) {
//     s_variance = rsqrtf(variance / hidden_size + epsilon);
//   }
//   __syncthreads();

//   for (int idx = threadIdx.x; idx < vec_hidden_size; idx += blockDim.x) {
//     int id = blockIdx.x * vec_hidden_size + idx;
//     _f16Vec<scalar_t, width> temp = residual_v[id];
//     temp *= s_variance;
//     temp *= weight_v[idx];
//     out_v_t temp_quant = fp8::scaled_vec_conversion<out_v_t, in_v_t>(
//         *reinterpret_cast<in_v_t*>(&temp), scale);
//     out_v[id] = temp_quant;
//   }
// }

/* Generic scaled_fused_add_rms_norm_kernel
   The width field is not used here but necessary for other specializations.
 */
// template <typename scalar_t, int width>
// __global__ std::enable_if_t<(width == 0) || !_typeConvert<scalar_t>::exists>
// scaled_fused_add_rms_norm_kernel(
//     c10::Float8_e4m3fnuz* __restrict__ out,  // [..., hidden_size]
//     scalar_t* __restrict__ input,            // [..., hidden_size]
//     scalar_t* __restrict__ residual,         // [..., hidden_size]
//     const scalar_t* __restrict__ weight,     // [hidden_size]
//     const float epsilon, const float scale, const int num_tokens,
//     const int hidden_size) {
//   __shared__ float s_variance;
//   float variance = 0.0f;

//   for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
//     scalar_t z = input[blockIdx.x * hidden_size + idx];
//     z += residual[blockIdx.x * hidden_size + idx];
//     float x = (float)z;
//     variance += x * x;
//     residual[blockIdx.x * hidden_size + idx] = z;
//   }

//   using BlockReduce = hipcub::BlockReduce<float, 1024>;
//   __shared__ typename BlockReduce::TempStorage reduceStore;
//   variance = BlockReduce(reduceStore).Reduce(variance, hipcub::Sum{}, blockDim.x);

//   if (threadIdx.x == 0) {
//     s_variance = rsqrtf(variance / hidden_size + epsilon);
//   }
//   __syncthreads();

//   for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
//     float x = (float)residual[blockIdx.x * hidden_size + idx];
//     float r = (x * s_variance) * (float)weight[idx] / scale;
//     out[blockIdx.x * hidden_size + idx] = c10::Float8_e4m3fnuz(
//         hip_fp8(r).data, c10::Float8_e4m3fnuz::from_bits());
//   }
// }

}  // namespace vllm

void rms_norm(torch::Tensor& out,     // [..., hidden_size]
              torch::Tensor& input,   // [..., hidden_size]
              torch::Tensor& weight,  // [hidden_size]
              double epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_kernel", [&] {
    vllm::rms_norm_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
  });
}

// void scaled_rms_norm(torch::Tensor& out,     // [..., hidden_size]
//                      torch::Tensor& input,   // [..., hidden_size]
//                      torch::Tensor& weight,  // [hidden_size]
//                      torch::Tensor& scale, double epsilon) {
//   int hidden_size = input.size(-1);
//   int num_tokens = input.numel() / hidden_size;

//   dim3 grid(num_tokens);
//   dim3 block(std::min(hidden_size, 1024));
//   const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
//   const hipStream_t stream = at::cuda::getCurrentCUDAStream();
//   VLLM_DISPATCH_FLOATING_TYPES(
//       input.scalar_type(), "scaled_rms_norm_kernel", [&] {
//         vllm::scaled_rms_norm_kernel<scalar_t><<<grid, block, 0, stream>>>(
//             out.data_ptr<c10::Float8_e4m3fnuz>(), input.data_ptr<scalar_t>(),
//             weight.data_ptr<scalar_t>(), 1.0 / (*scale.data_ptr<float>()),
//             epsilon, num_tokens, hidden_size);
//       });
// }

#define LAUNCH_FUSED_ADD_RMS_NORM(width)                                       \
  VLLM_DISPATCH_FLOATING_TYPES(                                                \
      input.scalar_type(), "fused_add_rms_norm_kernel", [&] {                  \
        vllm::fused_add_rms_norm_kernel<scalar_t, width>                       \
            <<<grid, block, 0, stream>>>(input.data_ptr<scalar_t>(),           \
                                         residual.data_ptr<scalar_t>(),        \
                                         weight.data_ptr<scalar_t>(), epsilon, \
                                         num_tokens, hidden_size);             \
      });

void fused_add_rms_norm(torch::Tensor& input,     // [..., hidden_size]
                        torch::Tensor& residual,  // [..., hidden_size]
                        torch::Tensor& weight,    // [hidden_size]
                        double epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  /* This kernel is memory-latency bound in many scenarios.
     When num_tokens is large, a smaller block size allows
     for increased block occupancy on CUs and better latency
     hiding on global mem ops. */
  const int max_block_size = (num_tokens < 256) ? 1024 : 256;
  dim3 block(std::min(hidden_size, max_block_size));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  /*If the tensor types are FP16/BF16, try to use the optimized kernel
    with packed + vectorized ops.
    Max optimization is achieved with a width-8 vector of FP16/BF16s
    since we can load at most 128 bits at once in a global memory op.
    However, this requires each tensor's data to be aligned to 16
    bytes.
   */
  auto inp_ptr = reinterpret_cast<std::uintptr_t>(input.data_ptr());
  auto res_ptr = reinterpret_cast<std::uintptr_t>(residual.data_ptr());
  auto wt_ptr = reinterpret_cast<std::uintptr_t>(weight.data_ptr());
  bool ptrs_are_aligned =
      inp_ptr % 16 == 0 && res_ptr % 16 == 0 && wt_ptr % 16 == 0;
  if (ptrs_are_aligned && hidden_size % 8 == 0) {
    LAUNCH_FUSED_ADD_RMS_NORM(8);
  } else {
    LAUNCH_FUSED_ADD_RMS_NORM(0);
  }
}

// #define LAUNCH_SCALED_FUSED_ADD_RMS_NORM(width)                            \
//   VLLM_DISPATCH_FLOATING_TYPES(                                            \
//       input.scalar_type(), "scaled_fused_add_rms_norm_kernel", [&] {       \
//         vllm::scaled_fused_add_rms_norm_kernel<scalar_t, width>            \
//             <<<grid, block, 0, stream>>>(                                  \
//                 out.data_ptr<c10::Float8_e4m3fnuz>(),                      \
//                 input.data_ptr<scalar_t>(), residual.data_ptr<scalar_t>(), \
//                 weight.data_ptr<scalar_t>(), epsilon,                      \
//                 *scale.data_ptr<float>(), num_tokens, hidden_size);        \
//       });

// void scaled_fused_add_rms_norm(torch::Tensor& out,       // [..., hidden_size]
//                                torch::Tensor& input,     // [..., hidden_size]
//                                torch::Tensor& residual,  // [..., hidden_size]
//                                torch::Tensor& weight,    // [hidden_size]
//                                torch::Tensor& scale, double epsilon) {
//   int hidden_size = input.size(-1);
//   int num_tokens = input.numel() / hidden_size;

//   dim3 grid(num_tokens);
//   /* This kernel is memory-latency bound in many scenarios.
//      When num_tokens is large, a smaller block size allows
//      for increased block occupancy on CUs and better latency
//      hiding on global mem ops. */
//   const int max_block_size = (num_tokens < 256) ? 1024 : 256;
//   dim3 block(std::min(hidden_size, max_block_size));
//   const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
//   const hipStream_t stream = at::cuda::getCurrentCUDAStream();
//   /*If the tensor types are FP16/BF16, try to use the optimized kernel
//     with packed + vectorized ops.
//     Max optimization is achieved with a width-8 vector of FP16/BF16s
//     since we can load at most 128 bits at once in a global memory op.
//     However, this requires each tensor's data to be aligned to 16
//     bytes.
//    */
//   auto inp_ptr = reinterpret_cast<std::uintptr_t>(input.data_ptr());
//   auto res_ptr = reinterpret_cast<std::uintptr_t>(residual.data_ptr());
//   auto wt_ptr = reinterpret_cast<std::uintptr_t>(weight.data_ptr());
//   bool ptrs_are_aligned =
//       inp_ptr % 16 == 0 && res_ptr % 16 == 0 && wt_ptr % 16 == 0;
//   if (ptrs_are_aligned && hidden_size % 8 == 0) {
//     LAUNCH_SCALED_FUSED_ADD_RMS_NORM(8);
//   } else {
//     LAUNCH_SCALED_FUSED_ADD_RMS_NORM(0);
//   }
// }

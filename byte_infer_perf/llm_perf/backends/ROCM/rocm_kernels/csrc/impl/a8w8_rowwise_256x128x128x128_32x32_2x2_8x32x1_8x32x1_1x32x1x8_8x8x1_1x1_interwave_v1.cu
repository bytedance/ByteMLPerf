// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.

#include "gemm_a8w8_common.cuh"

template <typename DDataType, typename EDataType = DDataType>
torch::Tensor
a8w8_rowwise_256x128x128x128_32x32_2x2_8x32x1_8x32x1_1x32x1x8_8x8x1_1x1_interwave_v1(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y)
{
    // A kernel that seems to work well on mid sized tensors.

    // Check if this input needs to be padded.
    int M = size_to_dim_(XQ.dim() - 1, XQ.sizes());
    int N = WQ.size(0);
    int K = WQ.size(1);
    bool pad = (K % 128 != 0);

    // Dispatch based on whether padding is needed or not.
    if (pad)
    {
        using DeviceGemmInstance = DeviceGemmHelper<
            DDataType, EDataType,
            256,
            128,
            128,
            128,
            32,
            32,
            2,
            2,
            S<8, 32, 1>,
            S<8, 32, 1>,
            S<1, 32, 1, 8>,
            S<8, 8, 1>,
            1,
            1,
            ck::BlockGemmPipelineScheduler::Interwave,
            ck::BlockGemmPipelineVersion::v1,
            ck::tensor_operation::device::GemmSpecialization::KPadding>;
        // Run kernel instance.
        return gemm_a8w8_rowwise_impl<DDataType, EDataType, DeviceGemmInstance>(
            XQ, WQ, x_scale, w_scale, Y);
    }
    else
    {
        using DeviceGemmInstance = DeviceGemmHelper<
            DDataType, EDataType,
            256,
            128,
            128,
            128,
            32,
            32,
            2,
            2,
            S<8, 32, 1>,
            S<8, 32, 1>,
            S<1, 32, 1, 8>,
            S<8, 8, 1>,
            1,
            1,
            ck::BlockGemmPipelineScheduler::Interwave,
            ck::BlockGemmPipelineVersion::v1,
            ck::tensor_operation::device::GemmSpecialization::Default>;
        // Run kernel instance.
        return gemm_a8w8_rowwise_impl<DDataType, EDataType, DeviceGemmInstance>(
            XQ, WQ, x_scale, w_scale, Y);
    }
}

template torch::Tensor
a8w8_rowwise_256x128x128x128_32x32_2x2_8x32x1_8x32x1_1x32x1x8_8x8x1_1x1_interwave_v1<F16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);

template torch::Tensor
a8w8_rowwise_256x128x128x128_32x32_2x2_8x32x1_8x32x1_1x32x1x8_8x8x1_1x1_interwave_v1<B16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);

template torch::Tensor
a8w8_rowwise_256x128x128x128_32x32_2x2_8x32x1_8x32x1_1x32x1x8_8x8x1_1x1_interwave_v1<F32, F16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);

template torch::Tensor
a8w8_rowwise_256x128x128x128_32x32_2x2_8x32x1_8x32x1_1x32x1x8_8x8x1_1x1_interwave_v1<F32, B16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);
// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.

#include "gemm_a8w8_common.cuh"

template <typename DDataType, typename EDataType = DDataType>
torch::Tensor
a8w8_rowwise_128x16x32x128_16x16_1x1_4x64x1_4x64x1_1x32x1x8_8x8x1_1x1_intrawave_v2(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y)
{
    // The smallest kernel we have available. Works well for memory bound shapes.
    using DeviceGemmInstance = DeviceGemmHelper<
        DDataType, EDataType,
        128,
        16,
        32,
        128,
        16,
        16,
        1,
        1,
        S<8, 16, 1>,
        S<8, 16, 1>,
        S<1, 16, 1, 8>,
        S<4, 4, 1>,
        1,
        1,
        ck::BlockGemmPipelineScheduler::Interwave,
        ck::BlockGemmPipelineVersion::v2,
        ck::tensor_operation::device::GemmSpecialization::Default>;
    // Run kernel instance.
    return gemm_a8w8_rowwise_impl<DDataType, EDataType, DeviceGemmInstance>(XQ, WQ, x_scale, w_scale, Y);
}

template torch::Tensor
a8w8_rowwise_128x16x32x128_16x16_1x1_4x64x1_4x64x1_1x32x1x8_8x8x1_1x1_intrawave_v2<F16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);

template torch::Tensor
a8w8_rowwise_128x16x32x128_16x16_1x1_4x64x1_4x64x1_1x32x1x8_8x8x1_1x1_intrawave_v2<B16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);

template torch::Tensor
a8w8_rowwise_128x16x32x128_16x16_1x1_4x64x1_4x64x1_1x32x1x8_8x8x1_1x1_intrawave_v2<F32, F16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);

template torch::Tensor
a8w8_rowwise_128x16x32x128_16x16_1x1_4x64x1_4x64x1_1x32x1x8_8x8x1_1x1_intrawave_v2<F32, B16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);
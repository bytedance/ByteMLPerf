// SPDX-License-Identifier: MIT
// Copyright (c) 2024, Advanced Micro Devices, Inc. All rights reserved.

#include "gemm_a8w8_common.cuh"

template <typename DDataType, typename EDataType = DDataType>
torch::Tensor
a8w8_rowwise_128x32x16x128_16x16_1x1_8x16x1_8x16x1_1x16x1x8_2x2x1_1x1_interwave_v2(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y)
{
    // A small kernel for small but not tiny shapes.

    // Check if this input needs to be padded.
    int M = size_to_dim_(XQ.dim() - 1, XQ.sizes());
    int N = WQ.size(0);
    int K = WQ.size(1);
    bool pad = (M % 32 != 0) || (N % 16 != 0) || (K % 128 != 0);

    if (pad)
    {
        using DeviceGemmInstance = DeviceGemmHelper<
            DDataType, EDataType,
            128,
            32,
            16,
            128,
            16,
            16,
            1,
            1,
            S<8, 16, 1>,
            S<8, 16, 1>,
            S<1, 16, 1, 8>,
            S<2, 2, 1>,
            1,
            1,
            ck::BlockGemmPipelineScheduler::Interwave,
            ck::BlockGemmPipelineVersion::v2>;
        // Run kernel instance.
        return gemm_a8w8_rowwise_impl<DDataType, EDataType, DeviceGemmInstance>(XQ, WQ, x_scale, w_scale, Y);
    }
    else
    {
        using DeviceGemmInstance = DeviceGemmHelper<
            DDataType, EDataType,
            128,
            32,
            16,
            128,
            16,
            16,
            1,
            1,
            S<8, 16, 1>,
            S<8, 16, 1>,
            S<1, 16, 1, 8>,
            S<2, 2, 1>,
            1,
            1,
            ck::BlockGemmPipelineScheduler::Interwave,
            ck::BlockGemmPipelineVersion::v2,
            ck::tensor_operation::device::GemmSpecialization::Default>;
        // Run kernel instance.
        return gemm_a8w8_rowwise_impl<DDataType, EDataType, DeviceGemmInstance>(XQ, WQ, x_scale, w_scale, Y);
    }
}

template torch::Tensor
a8w8_rowwise_128x32x16x128_16x16_1x1_8x16x1_8x16x1_1x16x1x8_2x2x1_1x1_interwave_v2<F16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);

template torch::Tensor
a8w8_rowwise_128x32x16x128_16x16_1x1_8x16x1_8x16x1_1x16x1x8_2x2x1_1x1_interwave_v2<B16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);

template torch::Tensor
a8w8_rowwise_128x32x16x128_16x16_1x1_8x16x1_8x16x1_1x16x1x8_2x2x1_1x1_interwave_v2<F32, F16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);

template torch::Tensor
a8w8_rowwise_128x32x16x128_16x16_1x1_8x16x1_8x16x1_1x16x1x8_2x2x1_1x1_interwave_v2<F32, B16>(
    torch::Tensor &XQ,
    torch::Tensor &WQ,
    torch::Tensor &x_scale,
    torch::Tensor &w_scale,
    torch::Tensor &Y);